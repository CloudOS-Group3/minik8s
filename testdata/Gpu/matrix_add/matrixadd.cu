#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define M 1000 // 矩阵的行数
#define N 1000 // 矩阵的列数

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // 用于将二维索引转换为一维索引

// 矩阵加法核函数
__global__ void matrixAddKernel(float *A, float *B, float *C, int rows, int cols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // 程块中的线程数量 * 程序块索引 + 块内线程索引
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < rows && idy < cols) {
        int index = IDX2C(idx, idy, rows);
        C[index] = A[index] + B[index]; // 计算矩阵 C 的元素
    }
}

int main(void) {
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float *devPtrA, *devPtrB, *devPtrC;
    float *a = 0, *b = 0, *c = 0;
    int i, j;

    // 在主机上分配内存
    a = (float *)malloc(M * N * sizeof(*a));
    b = (float *)malloc(M * N * sizeof(*b));
    c = (float *)malloc(M * N * sizeof(*c));

    if (!a || !b || !c) {
        printf("Host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // 初始化矩阵 A 和 B
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
            b[IDX2C(i,j,M)] = (float)(i * M + j + 1) * 2;
        }
    }

    // 在设备上分配内存
    cudaStat = hipMalloc((void**)&devPtrA, M * N * sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for A failed\n");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc((void**)&devPtrB, M * N * sizeof(*b));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for B failed\n");
        hipFree(devPtrA);
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc((void**)&devPtrC, M * N * sizeof(*c));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for C failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        return EXIT_FAILURE;
    }

    // 创建 cuBLAS 句柄
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return EXIT_FAILURE;
    }

    // 将矩阵 A 和 B 复制到设备
    stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("Data download for A failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix(M, N, sizeof(*b), b, M, devPtrB, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("Data download for B failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // 配置 CUDA 核函数的执行参数
    dim3 threadsPerBlock(16, 16); // 16x16的线程块, 共256个线程
    dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用矩阵加法核函数
    matrixAddKernel<<<blocksPerGrid, threadsPerBlock>>>(devPtrA, devPtrB, devPtrC, M, N);
    cudaStat = hipDeviceSynchronize();
    if (cudaStat != hipSuccess) {
        printf("Kernel execution failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // 将结果矩阵 C 从设备复制回主机
    stat = hipblasGetMatrix(M, N, sizeof(*c), devPtrC, M, c, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("Data upload for C failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // 释放设备内存和 cuBLAS 句柄
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    hipblasDestroy(handle);

    // 打印部分结果矩阵
    printf("Result matrix C (partial):\n");
    for (j = 0; j < (N < 10 ? N : 10); j++) {
        for (i = 0; i < (M < 10 ? M : 10); i++) {
            printf("%7.0f", c[IDX2C(i,j,M)]);
        }
        printf("\n");
    }

    // 释放主机内存
    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;
}
