#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define M 500 // 矩阵 A 的行数
#define K 500 // 矩阵 A 的列数和矩阵 B 的行数
#define N 500 // 矩阵 B 的列数

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void matrixMulKernel(float *A, float *B, float *C, int m, int k, int n) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += A[IDX2C(row, i, m)] * B[IDX2C(i, col, k)];
        }
        C[IDX2C(row, col, m)] = sum;
    }
}

int main(void) {
    hipError_t cudaStat;
    float *devPtrA, *devPtrB, *devPtrC;
    float *a = 0, *b = 0, *c = 0;
    int i, j;

    // 在主机上分配内存
    a = (float *)malloc(M * K * sizeof(*a));
    b = (float *)malloc(K * N * sizeof(*b));
    c = (float *)malloc(M * N * sizeof(*c));

    if (!a || !b || !c) {
        printf("Host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // 初始化矩阵 A 和 B
    for (j = 0; j < K; j++) {
        for (i = 0; i < M; i++) {
            a[IDX2C(i, j, M)] = (float)(rand() % 10); // 用随机值初始化矩阵 A
        }
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < K; i++) {
            b[IDX2C(i, j, K)] = (float)(rand() % 10); // 用随机值初始化矩阵 B
        }
    }

    // 在设备上分配内存
    cudaStat = hipMalloc((void**)&devPtrA, M * K * sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for A failed\n");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc((void**)&devPtrB, K * N * sizeof(*b));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for B failed\n");
        hipFree(devPtrA);
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc((void**)&devPtrC, M * N * sizeof(*c));
    if (cudaStat != hipSuccess) {
        printf("Device memory allocation for C failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        return EXIT_FAILURE;
    }

    // 将矩阵 A 和 B 复制到设备
    cudaStat = hipMemcpy(devPtrA, a, M * K * sizeof(*a), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        printf("Data transfer for A failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return EXIT_FAILURE;
    }
    cudaStat = hipMemcpy(devPtrB, b, K * N * sizeof(*b), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        printf("Data transfer for B failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return EXIT_FAILURE;
    }

    // 配置 CUDA 核函数的执行参数
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用矩阵乘法核函数
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(devPtrA, devPtrB, devPtrC, M, K, N);
    cudaStat = hipDeviceSynchronize();
    if (cudaStat != hipSuccess) {
        printf("Kernel execution failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return EXIT_FAILURE;
    }

    // 将结果矩阵 C 从设备复制回主机
    cudaStat = hipMemcpy(c, devPtrC, M * N * sizeof(*c), hipMemcpyDeviceToHost);
    if (cudaStat != hipSuccess) {
        printf("Data transfer for C failed\n");
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        return EXIT_FAILURE;
    }

    // 释放设备内存
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);

    // 打印部分结果矩阵
    printf("Result matrix C (partial):\n");
    for (j = 0; j < (N < 10 ? N : 10); j++) {
        for (i = 0; i < (M < 10 ? M : 10); i++) {
            printf("%7.0f", c[IDX2C(i,j,M)]);
        }
        printf("\n");
    }

    // 释放主机内存
    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;
}
